#include "hip/hip_runtime.h"
#include "misc.h"

#include <cstdint>
#include <omp.h>

#include "reduction.cuh"
#include "src/utils/cuda_utils.h"
#include "utils.h"

namespace NeuroFrame::Backend::CUDA {

__device__ int32_t answer;

// grid: [batch_size]
// block: [num_classes]
template<typename T>
__global__ void get_correct_sample_count_kernel(
	const T* __restrict__ pred_output,			// [batch_size, num_classes]
	const int32_t* __restrict__ ground_truth,	// [batch_size]
	int64_t batch_size,
	int64_t num_classes
) {
	int64_t batch_id = blockIdx.x;
	int64_t class_id = threadIdx.x;
	T my_pred_output = pred_output[INDEX_2D(batch_size, num_classes, batch_id, class_id)];

	T group_max_pred_output = block_reduce_max_broadcast<double>(my_pred_output);
	if (group_max_pred_output == my_pred_output && ground_truth[batch_id] == class_id) {
		atomicAdd(&answer, 1);
	}
}

int64_t get_correct_sample_count(const Tensor &pred_output, const Tensor &ground_truth) {
	int64_t batch_size = pred_output.shape[0];
	int64_t num_classes = pred_output.shape[1];

	int32_t* answer_dev_addr;
	hipGetSymbolAddress((void**)&answer_dev_addr, answer);

	CUDA_CHECK(hipMemset(answer_dev_addr, 0, sizeof(int32_t)));
	DISPATCH_ON_DTYPE_CUDA_BACKEND(
		pred_output.dtype,
		get_correct_sample_count_kernel<<<batch_size, num_classes>>>(
			(const T*) pred_output.data_ptr(),
			(const int32_t*) ground_truth.data_ptr(),
			batch_size,
			num_classes
		)
	);

	int32_t answer_h;
	CUDA_CHECK(hipMemcpyFromSymbol(&answer_h, HIP_SYMBOL(answer), sizeof(int32_t)));

	return answer_h;
}

}

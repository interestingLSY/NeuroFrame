#include "hip/hip_runtime.h"
#include "misc.h"

#include <cstdint>
#include <omp.h>

#include "reduction.cuh"
#include "src/utils/cuda_utils.h"
#include "utils.h"

namespace NeuroFrame::Backend::CUDA {

__device__ int32_t answer;

// grid: [batch_size]
// block: [num_classes]
template<typename T>
__global__ void get_correct_sample_count_kernel(
	const T* __restrict__ pred_output,			// [batch_size, num_classes]
	const int32_t* __restrict__ ground_truth,	// [batch_size]
	int64_t batch_size,
	int64_t num_classes
) {
	int64_t batch_id = blockIdx.x;
	int64_t class_id = threadIdx.x;
	T my_pred_output = pred_output[INDEX_2D(batch_size, num_classes, batch_id, class_id)];

	T group_max_pred_output = block_reduce_max_broadcast<double>(my_pred_output);
	if (group_max_pred_output == my_pred_output && ground_truth[batch_id] == class_id) {
		atomicAdd(&answer, 1);
	}
}

int64_t get_correct_sample_count(const Tensor &pred_output, const Tensor &ground_truth) {
	int64_t batch_size = pred_output.shape[0];
	int64_t num_classes = pred_output.shape[1];

	int32_t* answer_dev_addr;
	hipGetSymbolAddress((void**)&answer_dev_addr, answer);

	CUDA_CHECK(hipMemset(answer_dev_addr, 0, sizeof(int32_t)));
	DISPATCH_ON_DTYPE_CUDA_BACKEND(
		pred_output.dtype,
		get_correct_sample_count_kernel<<<batch_size, num_classes>>>(
			(const T*) pred_output.data_ptr(),
			(const int32_t*) ground_truth.data_ptr(),
			batch_size,
			num_classes
		)
	);

	int32_t answer_h;
	CUDA_CHECK(hipMemcpyFromSymbol(&answer_h, HIP_SYMBOL(answer), sizeof(int32_t)));

	return answer_h;
}

template<typename T, bool HAVE_MOMENTUM, bool HAVE_WEIGHT_DECAY>
__global__ void sgd_grad_update_kernel(
	T* __restrict__ weight,
	const T* __restrict__ grad,
	T* __restrict__ momentum,
	T learning_rate,
	T momentum_factor,
	T weight_decay,
	int64_t num_elements
) {
	#pragma unroll 2
	for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < num_elements; i += blockDim.x * gridDim.x) {
		T cur_grad = HAVE_WEIGHT_DECAY ? grad[i] + weight_decay * weight[i] : grad[i];
		if constexpr (HAVE_MOMENTUM) {
			cur_grad = momentum[i] = momentum_factor * momentum[i] + cur_grad;
		}
		weight[i] = weight[i] - learning_rate*cur_grad;
	}
}

void sgd_grad_update(Tensor &weight, const Tensor &grad, Tensor &momentum, double learning_rate, double momentum_factor, double weight_decay) {
	int64_t numel = weight.numel();
	int64_t block_size = ELEMENT_WISE_KERNEL_BLOCK_SIZE;
	int64_t grid_size = element_wise_kernel_get_num_grids(numel);
	#define DISPATCH(have_momentum, have_weight_decay) \
		DISPATCH_ON_DTYPE_CUDA_BACKEND(weight.dtype, \
			sgd_grad_update_kernel<T, have_momentum, have_weight_decay><<<grid_size, block_size>>>( \
				(T*) weight.data_ptr(), \
				(const T*) grad.data_ptr(), \
				(T*) momentum.data_ptr(), \
				(T) learning_rate, \
				(T) momentum_factor, \
				(T) weight_decay, \
				numel \
			));

	if (momentum_factor == 0) {
		if (weight_decay == 0) {
			DISPATCH(false, false);
		} else {
			DISPATCH(false, true);
		}
	} else {
		if (weight_decay == 0) {
			DISPATCH(true, false);
		} else {
			DISPATCH(true, true);
		}
	}
}

}

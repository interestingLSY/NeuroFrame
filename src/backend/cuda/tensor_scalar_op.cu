#include "hip/hip_runtime.h"
#include "tensor_scalar_op.h"

#include <stdexcept>
#include <hip/hip_runtime.h>
#include <cstdlib>

#include "src/tensor/tensor.h"
#include "src/basic/log.h"
#include "src/basic/scalar.h"
#include "utils.h"

namespace NeuroFrame::Backend::CUDA {

enum class TENSOR_SCALAR_OP_TYPE {
	ADDS
};

template <typename T, TENSOR_SCALAR_OP_TYPE OP_TYPE>
__device__ __forceinline__ T perform_tensor_scalar_op(const T &tensor_elem, const T &scalar_val) {
	if constexpr (OP_TYPE == TENSOR_SCALAR_OP_TYPE::ADDS) {
		return tensor_elem + scalar_val;
	}
}

template <typename T, TENSOR_SCALAR_OP_TYPE OP_TYPE>
__global__ void tensor_scalar_op_kernel(
	T *output,
	const T *input,
	const T &scalar_val,
	int64_t n
) {
	#pragma unroll 4
	for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
		output[i] = perform_tensor_scalar_op<T, OP_TYPE>(input[i], scalar_val);
	}
}

constexpr int64_t BLOCK_SIZE = 256;
const auto GRID_SIZE_CALCULATOR = [](int64_t n) {
	return std::min((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (int64_t)16384);
};

#define DEFINE_TENSOR_SCALAR_OP(NAME, OP_TYPE) \
Tensor NAME(const Tensor &input, const Scalar &scalar) {\
	int64_t n = input.numel(); \
	Tensor output(input.shape, input.dtype, input.device); \
	int64_t block_size = BLOCK_SIZE; \
	int64_t grid_size = GRID_SIZE_CALCULATOR(block_size); \
	DISPATCH_ON_DTYPE_CUDA_BACKEND(input.dtype,  \
		tensor_scalar_op_kernel<T, OP_TYPE><<<grid_size, block_size>>>( \
			(T*) output.data_ptr(), \
			(const T*) input.data_ptr(), \
			scalar.to_c_dtype<T>(), \
			n \
		) \
	); \
	return output; \
}

DEFINE_TENSOR_SCALAR_OP(tensor_adds, TENSOR_SCALAR_OP_TYPE::ADDS)


}
#include "hip/hip_runtime.h"
#include "pool.h"

#include <stdexcept>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "src/tensor/tensor.h"
#include "src/basic/log.h"
#include "utils.h"
#include "cudnn_utils.h"

namespace NeuroFrame::Backend::CUDA {

Tensor pool_forward(const Tensor &input, int pool_size, int stride, int padding) {
	if (input.dim() != 4) {
		LOG_FATAL("Input tensor must have 4 dimensions");
	}

	int64_t n = input.shape[0];
	int64_t c = input.shape[1];
	int64_t h = input.shape[2];
	int64_t w = input.shape[3];
	int out_n, out_c, out_h, out_w;
	
	static hipdnnPoolingDescriptor_t pool_desc;
	static hipdnnTensorDescriptor_t input_desc, output_desc;
	static bool desc_inited = false;
	if (!desc_inited) {
		CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&pool_desc));
		CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
		CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
		desc_inited = true;
	}
	CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
		input_desc,
		HIPDNN_TENSOR_NCHW,
		get_cudnn_data_type(input.dtype),
		n,
		c,
		h,
		w
	));
	CUDNN_CHECK(hipdnnSetPooling2dDescriptor(
		pool_desc,
		HIPDNN_POOLING_MAX,
		HIPDNN_NOT_PROPAGATE_NAN,
		pool_size,
		pool_size,
		padding,
		padding,
		stride,
		stride
	));
	CUDNN_CHECK(hipdnnGetPooling2dForwardOutputDim(
		pool_desc,
		input_desc,
		&out_n,
		&out_c,
		&out_h,
		&out_w
	));
	CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
		output_desc,
		HIPDNN_TENSOR_NCHW,
		get_cudnn_data_type(input.dtype),
		out_n,
		out_c,
		out_h,
		out_w
	));
	assert(out_n == n);
	assert(out_c == c);

	std::vector<int64_t> output_shape = {n, c, out_h, out_w};
	Tensor output(output_shape, input.dtype, input.device);

	auto [alpha_ptr, beta_ptr] = get_alpha_beta_ptrs(input.dtype);

	CUDNN_CHECK(hipdnnPoolingForward(
		cudnn_handle,
		pool_desc,
		alpha_ptr,
		input_desc,
		input.data_ptr(),
		beta_ptr,
		output_desc,
		output.data_ptr()
	));

	return output;
}

Tensor pool_backward(const Tensor &output_grad, const Tensor &input, const Tensor &output, int pool_size, int stride, int padding) {
	static hipdnnPoolingDescriptor_t pool_desc;
	static hipdnnTensorDescriptor_t input_desc, output_desc;	// input_desc and input_grad_desc are the same thing
	static bool desc_inited = false;
	if (!desc_inited) {
		CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&pool_desc));
		CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
		CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
		desc_inited = true;
	}

	int64_t n = input.shape[0];
	int64_t c = input.shape[1];
	int64_t h = input.shape[2];
	int64_t w = input.shape[3];
	int64_t out_h = output.shape[2];
	int64_t out_w = output.shape[3];

	CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
		input_desc,
		HIPDNN_TENSOR_NCHW,
		get_cudnn_data_type(input.dtype),
		n, c, h, w
	));
	CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
		output_desc,
		HIPDNN_TENSOR_NCHW,
		get_cudnn_data_type(output.dtype),
		n, c, out_h, out_w
	));
	CUDNN_CHECK(hipdnnSetPooling2dDescriptor(
		pool_desc,
		HIPDNN_POOLING_MAX,
		HIPDNN_NOT_PROPAGATE_NAN,
		pool_size,
		pool_size,
		padding,
		padding,
		stride,
		stride
	));

	std::vector<int64_t> input_grad_shape = {n, c, h, w};
	Tensor input_grad(input_grad_shape, input.dtype, input.device);

	auto [alpha_ptr, beta_ptr] = get_alpha_beta_ptrs(input.dtype);

	CUDNN_CHECK(hipdnnPoolingBackward(
		cudnn_handle,
		pool_desc,
		alpha_ptr,
		output_desc,
		output.data_ptr(),
		output_desc,
		output_grad.data_ptr(),
		input_desc,
		input.data_ptr(),
		beta_ptr,
		input_desc,
		input_grad.data_ptr()
	));

	return input_grad;
}

}